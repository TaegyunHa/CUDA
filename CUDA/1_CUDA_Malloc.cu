#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "utils.h"

#include <hip/hip_runtime.h>
#include ""

// #define ARRAY_1D
#ifdef ARRAY_1D

// Device Code
__global__ void VecAdd(float* a, float* b, float* c, int N)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid < N)
		c[gid] = a[gid] + b[gid];
}

int main()
{
	int N = 64;
	size_t byteSize = sizeof(float) * N;

	// Allocate input vectors in host memory
	float* h_A = (float*)malloc(byteSize);
	float* h_B = (float*)malloc(byteSize);
	float* h_C = (float*)malloc(byteSize);

	// Init input vectors
	utils::initVec<float>(h_A, 2, N);
	utils::initVec<float>(h_B, 3, N);

	// Allocate vectors in device memory
	float* d_A;
	hipMalloc(&d_A, byteSize);
	float* d_B;
	hipMalloc(&d_B, byteSize);
	float* d_C;
	hipMalloc(&d_C, byteSize);

	// Copy vectors from host to device memory
	hipMemcpy(d_A, h_A, byteSize, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_A, byteSize, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadPerBlock	= 256; // 16*16
	int blocksPerGrid	= (N + threadPerBlock - 1) / threadPerBlock;
	// This is done to make a margine to avoid floor result by
	// int-wise division. For example, 1/2 = 0
	VecAdd << <blocksPerGrid, threadPerBlock >> > (d_A, d_B, d_C, N);
	

	// Copy result from device to host memory
	hipMemcpy(h_C, d_C, byteSize, hipMemcpyDeviceToHost);
	
	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// Display result
	utils::printVec<float>(h_C, N);

	// Free host memory
	free(h_A);
	free(h_B);
	free(h_C);

	//hipDeviceSynchronize();
	hipDeviceReset();
	return 0;
}
#endif // ARRAY_1D


#define ARRAY_2D
#ifdef ARRAY_2D

// Device Code
__global__ void loop2DArray(float* d_pArray, size_t pitch, int width, int height)
{
	for (int rowIdx = 0; rowIdx < height; ++rowIdx)
	{
		// (char*) is for 1byte increment
		float* row = (float*)((char*)d_pArray + rowIdx * pitch);
		for (int colIdx = 0; colIdx < width; ++colIdx)
		{
			// value will be corrupted when multi-threads try to access
			float& element = row[colIdx];
			printf("%f\n", ++element);
		}
	}
}

int main()
{
	int width = 64;
	int height = 64;
	float* d_pArray;
	float* d_pArrayDst;
	size_t pitch;
	size_t pitchDst;

	// Allocate 2D array in host memory
	size_t h_pitch	= sizeof(float) * width;
	size_t byteSize = sizeof(float) * width * height;
	float* h_pArray = (float*)malloc(byteSize);
	utils::initVec<float>(h_pArray, 1.f, width * height);
	utils::printVec<float>(h_pArray, width * height);

	// Allocate 2D array and get a pitch
	hipMallocPitch(&d_pArray, &pitch, sizeof(float) * width, height);
	hipMallocPitch(&d_pArrayDst, &pitchDst, sizeof(float) * width, height);

	// Copy array from host to device
	hipMemcpy2D(d_pArray, pitch, h_pArray, h_pitch, sizeof(float) * width, height, hipMemcpyHostToDevice);
	
	// Invoke kernel
	size_t N = 51200;
	int threadsPerBlock = 512;
	int blocksPerGrid = N / threadsPerBlock;	
	// loop2DArray << <blocksPerGrid, threadsPerBlock >> > (d_pArray, pitch, width, height);
	loop2DArray << <1, 1>> > (d_pArray, pitch, width, height);

	// Copy array from device to device
	hipMemcpy2D(d_pArrayDst, pitchDst, d_pArray, pitch, sizeof(float) * width, height, hipMemcpyDeviceToDevice);
	loop2DArray << <1, 1 >> > (d_pArrayDst, pitchDst, width, height);

	// Copy array from device to host
	hipMemcpy2D(h_pArray, h_pitch, d_pArrayDst, pitchDst, sizeof(float) * width, height, hipMemcpyDeviceToHost);
	utils::printVec(h_pArray, width * height);

	// free memory
	hipFree(d_pArray);
	hipFree(d_pArrayDst);
	free(h_pArray);

	hipDeviceReset();
	return 0;
}
#endif // ARRAY_2D


//#define ARRAY_3D
#ifdef ARRAY_3D

// Device code
__global__ void loop3DArray(hipPitchedPtr d_pPitch, int width, int height, int depth)
{
	char* d_ptr = (char*)d_pPitch.ptr;
	size_t pitch = d_pPitch.pitch;
	size_t slicePitch = pitch * height;

	for (int z = 0; z < depth; ++z)
	{
		char* slice = d_ptr + (z * slicePitch);
		for (int y = 0; y < height; ++y)
		{
			float* row = (float*)(slice + (y * pitch));
			for (int x = 0; x < width; ++x)
			{
				float element = row[x];
			}
		}
	}
}

int main()
{
	int width = 64;
	int height = 64;
	int depth = 64;
	hipExtent extent = make_hipExtent(sizeof(float) * width, height, depth);

	// Allocate 3D array
	hipPitchedPtr d_pPitch;
	hipMalloc3D(&d_pPitch, extent);
	
	// Invoke kernel
	loop3DArray << <100, 512 >> > (d_pPitch, width, height, depth);

	hipDeviceReset();
	return 0;
}
#endif // ARRAY_3D